// Copyright (c) 2018-2020, Michael P. Howard
// Copyright (c) 2021-2024, Auburn University
// Part of azplugins, released under the BSD 3-Clause License.

#include "DPDPotentials.cuh"

namespace azplugins
    {
namespace gpu
    {

//! Kernel driver for modified DPD potential
template hipError_t compute_dpd_potential<azplugins::detail::DPDEvaluatorGeneralWeight>(
    const dpd_pair_args_t& dpd_args,
    const typename azplugins::detail::DPDEvaluatorGeneralWeight::param_type* d_params);

    } // end namespace gpu
    } // end namespace azplugins
